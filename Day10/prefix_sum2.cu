
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <chrono>

# define SEGMENT_SIZE 32

// prefix sum - kogge stone algo
__global__ void prefixSum(float *mat, int length) {
    // create shared memory
    __shared__ float mat_s[SEGMENT_SIZE];
    // each thread is repsonsible for this index element into input array
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // intialise the shared mamory segment
    if (idx < length)
        mat_s[threadIdx.x] = mat[idx];
    else
        mat_s[threadIdx.x] = 0;

    for (int stride=1; stride<blockDim.x; stride*=2) {
        // wait for shared memory initialisation
        __syncthreads();
        float temp = 0.0f;
        // all threads with id < stride have already calculated their final value
        if(threadIdx.x >= stride)
            temp = mat_s[threadIdx.x] + mat_s[threadIdx.x - stride];
        // wait till all read operations are done into shared memory
        __syncthreads();
        // write the result back into shared memory
        if(threadIdx.x >= stride)
            mat_s[threadIdx.x] = temp;
    }
    // write result back into input array
    if (idx < length)
        mat[idx] = mat_s[threadIdx.x];
}

// prefix sum - kogge stone algo
__global__ void prefixSum2(float *mat, int length) {
    // create shared memory
    __shared__ float mat_s[SEGMENT_SIZE];
    // each thread is responsible for loading two elements into shared memory
    int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // load the element at index idx and idx + blockDim.x
    if (idx < length)
        mat_s[threadIdx.x] = mat[idx];
    if (idx + blockDim.x < length)
        mat_s[threadIdx.x + blockDim.x] = mat[idx + blockDim.x];

    // reduction
    for (int stride=1; stride<blockDim.x; stride*=2) {
        // wait for shared memory initialisation
        __syncthreads();
        int input_idx = (threadIdx.x + 1) * 2 * stride - 1;
        if (input_idx < SEGMENT_SIZE)
            mat_s[input_idx] += mat_s[input_idx - stride];
    }

    // distribution
    for (int stride=SEGMENT_SIZE/4; stride>0; stride /= 2) {
        // wait for shared memory initialisation
        __syncthreads();
        int input_idx = (threadIdx.x + 1) * 2 * stride - 1;
        if (input_idx < SEGMENT_SIZE)
            mat_s[input_idx + stride] += mat_s[input_idx];
    }

    // write result back into input array
    if (idx < length)
        mat[idx] = mat_s[threadIdx.x];
    if (threadIdx.x + blockDim.x < length)
        mat[idx + blockDim.x] = mat_s[threadIdx.x + blockDim.x];
}




int main() {
    // Define an input array of eight elements.
    const int length = 8;
    const int size = length * sizeof(float);
    float h_data[length] = {1, 2, 3, 4, 5, 6, 7, 8};

    // Allocate device memory.
    float *d_data;
    hipMalloc((void**)&d_data, size);

    // Copy input array from host to device.
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    // Launch kernel with one block. Even though SEGMENT_SIZE is 32, our input length is 8.
    int threadsPerBlock = SEGMENT_SIZE; // 32 threads per block
    int blocks = 1; // One block is enough because input length <= SEGMENT_SIZE.
    prefixSum2<<<blocks, threadsPerBlock>>>(d_data, length);

    // Wait for the kernel to complete.
    hipDeviceSynchronize();

    // Copy result from device to host.
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_data);

    // Print the prefix sum result.
    std::cout << "Prefix sum result: ";
    for (int i = 0; i < length; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
